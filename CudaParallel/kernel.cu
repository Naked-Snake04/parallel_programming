#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <windows.h>

using namespace std;

__global__ void matrixMult(const int* A, const int* B, int* C, int matrix)
{
    int i = matrix * (blockDim.y * blockIdx.y + threadIdx.y);
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int sum = 0;

    for (int k = 0; k < matrix; k++)
        sum += A[i + k] * B[k * matrix + j];

    int ind = matrix * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    C[ind] = sum;
}

int main(int argc, char** argv) {

    SetConsoleOutputCP(CP_UTF8);

    int threads, matrix;
    bool flag_thread = true, flag_matrix = true;
    string thread_input, matrix_size;

    cout << "Enter the number of threads: " << endl;

    while (flag_thread == true)
    {
        try
        {
            cin >> thread_input;
            threads = stoi(thread_input);

            if (threads < 0) 
            {
                cin.clear();
                cin.ignore(cin.rdbuf()->in_avail());
                cout << "Incorrect data entry. Try entering again" << endl;
                cout << "Enter the number of threads: ";
            }
            else
                flag_thread = false;
        }
        catch (invalid_argument)
        {
            cin.clear();
            cin.ignore(cin.rdbuf()->in_avail());
            cout << "Incorrect data entry. Try entering again" << endl;
            cout << "Enter the number of threads: ";
        }
    }

    cout << "Enter matrix size = ";

    while (flag_matrix == true)
    {
        try
        {
            cin >> matrix_size;
            matrix = stoi(matrix_size);

            if (matrix < 0)
            {
                cin.clear();
                cin.ignore(cin.rdbuf()->in_avail());
                cout << "Incorrect data entry. Try entering again" << endl;
                cout << "Enter the number of threads: ";
            }
            else
                flag_matrix = false;
        }
        catch (invalid_argument)
        {
            cin.clear();
            cin.ignore(cin.rdbuf()->in_avail());
            cout << "Incorrect data entry. Try entering again" << endl;
            cout << "Enter the number of threads: ";
        }
    }

    size_t size = matrix * matrix * sizeof(int);

    int* a = (int*)malloc(size);
    int* b = (int*)malloc(size);
    int* c = (int*)malloc(size);

    for (int i = 0; i < matrix; i++) {
        for (int j = 0; j < matrix; j++) {
            a[i * matrix + j] = i * j;
            b[i * matrix + j] = i * j;
        }
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int* calcA = NULL;
    hipMalloc((void**)&calcA, size);

    int* calcB = NULL;
    hipMalloc((void**)&calcB, size);

    int* calcC = NULL;
    hipMalloc((void**)&calcC, size);

    hipMemcpy(calcA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(calcB, b, size, hipMemcpyHostToDevice);

    cout << "Start programm" << endl;

    dim3 threadsPerBlock = dim3(threads, threads);
    dim3 blocksPerGrid = dim3(matrix / threads, matrix / threads);

    hipEventRecord(start, 0);
    matrixMult <<< blocksPerGrid, threadsPerBlock >>> (calcA, calcB, calcC, matrix);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);
    double time = kernelTime;
    cout << "Threads on block: " << threads
        << "; block in cell: " << matrix / threads
        << "; worktime: " << kernelTime / 1000 << " sec" << endl;
    hipMemcpy(c, calcC, size, hipMemcpyDeviceToHost);

    hipFree(calcA);
    hipFree(calcB);
    hipFree(calcC);
    free(a);
    free(b);
    free(c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
